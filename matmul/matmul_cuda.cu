#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <vector>
#include <mutex>

#include <hip/hip_runtime.h>

#include "ansi_codes.h"
#include "types.h"
#include "matmul_cuda.h"

#define checkCudaError(err) \
({ \
    __typeof__(err) _err = (err); \
    if (_err != hipSuccess) { \
        fprintf(stderr, "%s:%d: %s\n", __FILE__, __LINE__, hipGetErrorName(_err)); \
        exit(EXIT_FAILURE); \
    } \
})


/*
 * Guard against executing multiple CUDA streams at once.
 *
 * FIXME: For some reason we crash when we run from multiple threads.
 *        Figure out how to use the API concurrently.
 */
static std::mutex kernel_exec_mtx;

static std::vector<hipDeviceProp_t> cuda_devices;
static const hipDeviceProp_t& current_dev()
{
    assert(!cuda_devices.empty());
    return cuda_devices[0];
}

EXTERN_C int matmul_cu_init(bool verbose)
{
    int num_devices = 0;
    hipError_t err = hipGetDeviceCount(&num_devices);

    checkCudaError(err);

    if (num_devices <= 0) {
        printf("No cuda devices\n");
        return 1;
    }

    if (verbose)
        printf("num devices: %d\n", num_devices);

    for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
        auto &devprop = cuda_devices.emplace_back();
        checkCudaError(hipGetDeviceProperties(&devprop, dev_id));

        if (!verbose)
            continue;

        printf("name                : %s\n", devprop.name);
        printf("  processor count   : %d\n", devprop.multiProcessorCount);
        printf("  max blocks/proc   : %d\n", devprop.maxBlocksPerMultiProcessor);
        printf("  max threads/block : %d\n", devprop.maxThreadsPerBlock);
        printf("  max threads X     : %d\n", devprop.maxThreadsDim[0]);
        printf("  max threads Y     : %d\n", devprop.maxThreadsDim[1]);
        printf("  max threads Z     : %d\n", devprop.maxThreadsDim[2]);
        printf("  max grid X        : %d\n", devprop.maxGridSize[0]);
        printf("  max grid Y        : %d\n", devprop.maxGridSize[1]);
        printf("  max grid Z        : %d\n", devprop.maxGridSize[2]);
        printf("  warp size         : %d\n", devprop.warpSize);
        printf("  global memory     : %lu MB\n", devprop.totalGlobalMem / (1024UL * 1024UL));
        printf("  shared memory     : %lu KB\n", devprop.sharedMemPerBlock / 1024UL);
        printf("  L2                : %d KB\n", devprop.l2CacheSize / 1024);
        printf("  gpu clock         : %d MHz\n", devprop.clockRate / 1024);
        printf("  mem clock         : %d MHz\n", devprop.memoryClockRate / 1024);
        printf("  bus width         : %d\n", devprop.memoryBusWidth);
        putchar('\n');
    }

    return 0;
}

/*
 * We assume all matrcies are square and have the same dimensions.
 */
__global__ void kernel_matmul_cu(
    const i64 *lhs,
    const i64 *rhs,
          i64 *out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
        const u32 x = threadIdx.x + blockDim.x * blockIdx.x;
        const u32 y = threadIdx.y + blockDim.y * blockIdx.y;

        const bool out_of_bounds = x >= dim | y >= dim;
        if (out_of_bounds)
            return;

        out[x + y*out_stride] = 0;

        for (u32 i = 0; i < dim; ++i)
            out[x + y*out_stride] += lhs[i + y*lhs_stride] * rhs[x + i*rhs_stride];
}

/*
 * We assume all matrcies are square and have the same dimensions.
 */
__global__ void kernel_matmul_tiled_cu(
    const i64 *lhs,
    const i64 *rhs,
          i64 *out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
    constexpr u32 TILE_SIZE = 32 * 32;

    __shared__ i64 tilea[TILE_SIZE];
    __shared__ i64 tileb[TILE_SIZE];
    __shared__ i64 tilec[TILE_SIZE];

    /* Global coordinates for this thread. */
    const u32 gx = threadIdx.x + blockIdx.x * blockDim.x;
    const u32 gy = threadIdx.y + blockIdx.y * blockDim.y;

    const bool out_of_bounds = gx >= dim | gy >= dim;
    if (out_of_bounds)
        return;

    /* Local tile coordinates for this thread. */
    const u32 tx      = threadIdx.x;
    const u32 ty      = threadIdx.y;
    const u32 tstride = blockDim.x;

    tilec[tx + ty * tstride] = 0;

    /*
     * Row is a loop invariant for tilea.
     * Column is a loop invariant for tileb.
     */
    for (int j = 0; j < dim; j += blockDim.x) {

        /*
         * Load lhs and rhs tiles into shared memory.
         * Each thread loads one element.
         */
        tilea[tx + ty * tstride] = lhs[(tx+j) + gy * lhs_stride];
        tileb[tx + ty * tstride] = rhs[gx + (ty+j) * rhs_stride];

        /* Make sure tilea and tileb are populated. */
        __syncthreads();

        /* Accumulate results for current tile. */
        for (u32 i = 0; i < blockDim.x; ++i)
            tilec[tx + ty * tstride] += tilea[i + ty * tstride] * tileb[tx + i * tstride];

        /* Make sure we don't modify tilea and tileb before all threads are finihsed with per tile computation */
        __syncthreads();
    }

    out[gx + gy * out_stride] = tilec[tx + ty * tstride];
}

static int run_kernel_cu_umem_(
    const i64 *h_lhs,
    const i64 *h_rhs,
          i64 *h_out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
          i64 *u_lhs;
    const u32  lhs_size = dim * lhs_stride * sizeof(*u_lhs);

          i64 *u_rhs;
    const u32  rhs_size = dim * rhs_stride * sizeof(*u_rhs);

          i64 *u_out;
    const u32  out_size = dim * out_stride * sizeof(*u_out);

    checkCudaError(hipMallocManaged(&u_lhs, lhs_size));
    checkCudaError(hipMallocManaged(&u_rhs, rhs_size));
    checkCudaError(hipMallocManaged(&u_out, out_size));

    memcpy(u_lhs, h_lhs, lhs_size);
    memcpy(u_rhs, h_rhs, rhs_size);

    /* We assume block dim to be 32 */
    const u32 num_threads = std::min(dim, 32u);
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks = (dim + 31u) / 32u;
    const dim3 grid_dims(num_blocks, num_blocks);

    kernel_matmul_cu<<<grid_dims, block_dims>>>(
        u_lhs,
        u_rhs,
        u_out,
        dim,
        lhs_stride,
        rhs_stride,
        out_stride
    );
    hipDeviceSynchronize();

    memcpy(h_out, u_out, out_size);

    hipFree(u_out);
    hipFree(u_rhs);
    hipFree(u_lhs);

    checkCudaError(hipGetLastError());

    return 0;
}

static int run_kernel_cu_umem_tiled_(
    const i64 *h_lhs,
    const i64 *h_rhs,
          i64 *h_out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
          i64 *u_lhs;
    const u32  lhs_size = dim * lhs_stride * sizeof(*u_lhs);

          i64 *u_rhs;
    const u32  rhs_size = dim * rhs_stride * sizeof(*u_rhs);

          i64 *u_out;
    const u32  out_size = dim * out_stride * sizeof(*u_out);

    checkCudaError(hipMallocManaged(&u_lhs, lhs_size));
    checkCudaError(hipMallocManaged(&u_rhs, rhs_size));
    checkCudaError(hipMallocManaged(&u_out, out_size));

    memcpy(u_lhs, h_lhs, lhs_size);
    memcpy(u_rhs, h_rhs, rhs_size);

    /* We assume block dim to be 32 */
    const u32 num_threads = std::min(dim, 32u);
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks = (dim + num_threads - 1) / num_threads;
    const dim3 grid_dims(num_blocks, num_blocks);

    kernel_matmul_tiled_cu<<<grid_dims, block_dims>>>(
        u_lhs,
        u_rhs,
        u_out,
        dim,
        lhs_stride,
        rhs_stride,
        out_stride
    );
    hipDeviceSynchronize();

    memcpy(h_out, u_out, out_size);

    hipFree(u_out);
    hipFree(u_rhs);
    hipFree(u_lhs);

    checkCudaError(hipGetLastError());

    return 0;
}

EXTERN_C int run_kernel_cu(
          i64 *h_lhs,
    const u32  lhs_cols,
    const u32  lhs_rows,
    const u32  lhs_stride,

          i64 *h_rhs,
    const u32  rhs_cols,
    const u32  rhs_rows,
    const u32  rhs_stride,

          i64 *h_out,
    const u32  out_cols,
    const u32  out_rows,
    const u32  out_stride,

    cuda_kernel_variant variant
) {
    assert(lhs_cols == lhs_rows);
    assert(rhs_cols == rhs_rows);
    assert(out_cols == out_rows);
    assert(lhs_cols == rhs_cols);
    assert(lhs_cols == out_cols);

    if (strcmp(current_dev().name, "NVIDIA GeForce GTX 970")) {
        fprintf(stderr, "WARN: %s: kernel written for %s, but current device is %s.",
                __func__, "NVIDIA GeForce GTX 970", current_dev().name);
    }

    std::unique_lock lck(kernel_exec_mtx);

    switch (variant) {
    case cuda_kernel_variant::UMEM:
        return run_kernel_cu_umem_(
            h_lhs,
            h_rhs,
            h_out,
            lhs_cols,
            lhs_stride,
            rhs_stride,
            out_stride
        );

    case cuda_kernel_variant::UMEM_TILED:
        return run_kernel_cu_umem_tiled_(
            h_lhs,
            h_rhs,
            h_out,
            lhs_cols,
            lhs_stride,
            rhs_stride,
            out_stride
        );
    }

    __builtin_unreachable();
}

