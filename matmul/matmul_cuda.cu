#include "hip/hip_runtime.h"
/* vim: set tabstop=4 softtabstop=4 expandtab shiftwidth=4: */
#include <stdio.h>
#include <assert.h>

#include <array>
#include <atomic>
#include <chrono>
#include <vector>
#include <limits>
#include <mutex>

#include <hip/hip_runtime.h>

#include "ansi_codes.h"
#include "types.h"
#include "matmul_cuda.h"
#include "interrupt.h"

#include "print_utils.h"

#define checkCudaError(err) \
({ \
    __typeof__(err) _err = (err); \
    if (_err != hipSuccess) { \
        fprintf(stderr, "%s:%d: %s\n", __FILE__, __LINE__, hipGetErrorName(_err)); \
        exit(EXIT_FAILURE); \
    } \
})


/*
 * Guard against executing multiple CUDA streams at once.
 *
 * FIXME: For some reason we crash when we run from multiple threads.
 *        Figure out how to use the API concurrently.
 */
static std::mutex kernel_exec_mtx;

static std::vector<hipDeviceProp_t> cuda_devices;
static const hipDeviceProp_t& current_dev()
{
    assert(!cuda_devices.empty());
    return cuda_devices[0];
}

EXTERN_C int matmul_cu_init(bool verbose)
{
    int num_devices = 0;
    hipError_t err = hipGetDeviceCount(&num_devices);

    checkCudaError(err);

    if (num_devices <= 0) {
        printf("No cuda devices\n");
        return 1;
    }

    if (verbose)
        printf("num devices: %d\n", num_devices);

    for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
        auto &devprop = cuda_devices.emplace_back();
        checkCudaError(hipGetDeviceProperties(&devprop, dev_id));

        if (!verbose)
            continue;

        printf("name                : %s\n", devprop.name);
        printf("  arch              : %d.%d\n", devprop.major, devprop.minor);
        printf("  processor count   : %d\n", devprop.multiProcessorCount);
        printf("  max blocks/proc   : %d\n", devprop.maxBlocksPerMultiProcessor);
        printf("  max threads/block : %d\n", devprop.maxThreadsPerBlock);
        printf("  max threads X     : %d\n", devprop.maxThreadsDim[0]);
        printf("  max threads Y     : %d\n", devprop.maxThreadsDim[1]);
        printf("  max threads Z     : %d\n", devprop.maxThreadsDim[2]);
        printf("  max grid X        : %d\n", devprop.maxGridSize[0]);
        printf("  max grid Y        : %d\n", devprop.maxGridSize[1]);
        printf("  max grid Z        : %d\n", devprop.maxGridSize[2]);
        printf("  warp size         : %d\n", devprop.warpSize);
        printf("  global memory     : %lu MB\n", devprop.totalGlobalMem / (1024UL * 1024UL));
        printf("  shared memory     : %lu KB\n", devprop.sharedMemPerBlock / 1024UL);
        printf("  L2                : %d KB\n", devprop.l2CacheSize / 1024);
        printf("  gpu clock         : %d MHz\n", devprop.clockRate / 1024);
        printf("  mem clock         : %d MHz\n", devprop.memoryClockRate / 1024);
        printf("  bus width         : %d\n", devprop.memoryBusWidth);
        putchar('\n');
    }

    return 0;
}


/****************************************************************************
 * I64 Kernels
 ****************************************************************************/

/*
 * We assume all matrcies are square and have the same dimensions.
 */
__global__ void kernel_matmul_cu(
    const i64 *lhs,
    const i64 *rhs,
          i64 *out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
        const u32 x = threadIdx.x + blockDim.x * blockIdx.x;
        const u32 y = threadIdx.y + blockDim.y * blockIdx.y;

        const bool out_of_bounds = x >= dim | y >= dim;
        if (out_of_bounds)
            return;

        out[x + y*out_stride] = 0;

        for (u32 i = 0; i < dim; ++i)
            out[x + y*out_stride] += lhs[i + y*lhs_stride] * rhs[x + i*rhs_stride];
}

/*
 * We assume all matrcies are square and have the same dimensions.
 */
__global__ void kernel_matmul_tiled_cu(
    const i64 *lhs,
    const i64 *rhs,
          i64 *out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
    constexpr u32 TILE_SIZE = 32 * 32;

    __shared__ i64 tilea[TILE_SIZE];
    __shared__ i64 tileb[TILE_SIZE];
    __shared__ i64 tilec[TILE_SIZE];

    /* Global coordinates for this thread. */
    const u32 gx = threadIdx.x + blockIdx.x * blockDim.x;
    const u32 gy = threadIdx.y + blockIdx.y * blockDim.y;

    const bool out_of_bounds = gx >= dim | gy >= dim;
    if (out_of_bounds)
        return;

    /* Local tile coordinates for this thread. */
    const u32 tx      = threadIdx.x;
    const u32 ty      = threadIdx.y;
    const u32 tstride = blockDim.x;

    tilec[tx + ty * tstride] = 0;

    /*
     * Row is a loop invariant for tilea.
     * Column is a loop invariant for tileb.
     */
    for (int j = 0; j < dim; j += blockDim.x) {

        /*
         * Load lhs and rhs tiles into shared memory.
         * Each thread loads one element.
         */
        tilea[tx + ty * tstride] = lhs[(tx+j) + gy * lhs_stride];
        tileb[tx + ty * tstride] = rhs[gx + (ty+j) * rhs_stride];

        /* Make sure tilea and tileb are populated. */
        __syncthreads();

        /* Accumulate results for current tile. */
        for (u32 i = 0; i < blockDim.x; ++i)
            tilec[tx + ty * tstride] += tilea[i + ty * tstride] * tileb[tx + i * tstride];

        /* Make sure we don't modify tilea and tileb before all threads are finihsed with per tile computation */
        __syncthreads();
    }

    out[gx + gy * out_stride] = tilec[tx + ty * tstride];
}

/*
 * We assume all matrcies are square and have the same dimensions.
 */
__global__ void kernel_matmul_test_cu(
    const i64 *lhs,
    const i64 *rhs,
          i64 *out_,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
    /* Global coordinates for this thread. */
    const u32 gx = blockIdx.z + blockIdx.x * blockDim.x;
    const u32 gy = blockIdx.z + blockIdx.y * blockDim.y;

    const bool out_of_bounds = gx >= dim | gy >= dim;
    if (out_of_bounds)
        return;

    u32 out = 0;

    for (u32 i = 0; i < dim; ++i)
        out += lhs[i + gy * lhs_stride] * rhs[gx + i * rhs_stride];

    out_[gx + gy * out_stride] = out;
}

static int run_kernel_cu_umem_(
    const i64 *h_lhs,
    const i64 *h_rhs,
          i64 *h_out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
          i64 *u_lhs;
    const u32  lhs_size = dim * lhs_stride * sizeof(*u_lhs);

          i64 *u_rhs;
    const u32  rhs_size = dim * rhs_stride * sizeof(*u_rhs);

          i64 *u_out;
    const u32  out_size = dim * out_stride * sizeof(*u_out);

    checkCudaError(hipMallocManaged(&u_lhs, lhs_size));
    checkCudaError(hipMallocManaged(&u_rhs, rhs_size));
    checkCudaError(hipMallocManaged(&u_out, out_size));

    memcpy(u_lhs, h_lhs, lhs_size);
    memcpy(u_rhs, h_rhs, rhs_size);

    /* We assume block dim to be 32 */
    const u32 num_threads = std::min(dim, 32u);
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks = (dim + 31u) / 32u;
    const dim3 grid_dims(num_blocks, num_blocks);

    kernel_matmul_cu<<<grid_dims, block_dims>>>(
        u_lhs,
        u_rhs,
        u_out,
        dim,
        lhs_stride,
        rhs_stride,
        out_stride
    );
    hipDeviceSynchronize();

    memcpy(h_out, u_out, out_size);

    hipFree(u_out);
    hipFree(u_rhs);
    hipFree(u_lhs);

    checkCudaError(hipGetLastError());

    return 0;
}

static int run_kernel_cu_umem_tiled_(
    const i64 *h_lhs,
    const i64 *h_rhs,
          i64 *h_out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
          i64 *u_lhs;
    const u32  lhs_size = dim * lhs_stride * sizeof(*u_lhs);

          i64 *u_rhs;
    const u32  rhs_size = dim * rhs_stride * sizeof(*u_rhs);

          i64 *u_out;
    const u32  out_size = dim * out_stride * sizeof(*u_out);

    checkCudaError(hipMallocManaged(&u_lhs, lhs_size));
    checkCudaError(hipMallocManaged(&u_rhs, rhs_size));
    checkCudaError(hipMallocManaged(&u_out, out_size));

    memcpy(u_lhs, h_lhs, lhs_size);
    memcpy(u_rhs, h_rhs, rhs_size);

    /* We assume block dim to be 32 */
    const u32 num_threads = std::min(dim, 32u);
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks = (dim + num_threads - 1) / num_threads;
    const dim3 grid_dims(num_blocks, num_blocks);

    kernel_matmul_tiled_cu<<<grid_dims, block_dims>>>(
        u_lhs,
        u_rhs,
        u_out,
        dim,
        lhs_stride,
        rhs_stride,
        out_stride
    );
    hipDeviceSynchronize();

    memcpy(h_out, u_out, out_size);

    hipFree(u_out);
    hipFree(u_rhs);
    hipFree(u_lhs);

    checkCudaError(hipGetLastError());

    return 0;
}

static int run_kernel_cu_tiled_(
    const i64 *h_lhs,
    const i64 *h_rhs,
          i64 *h_out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
          i64 *d_lhs;
    const u32  lhs_size = dim * lhs_stride * sizeof(*d_lhs);

          i64 *d_rhs;
    const u32  rhs_size = dim * rhs_stride * sizeof(*d_rhs);

          i64 *d_out;
    const u32  out_size = dim * out_stride * sizeof(*d_out);

    checkCudaError(hipMalloc(&d_lhs, lhs_size));
    checkCudaError(hipMalloc(&d_rhs, rhs_size));
    checkCudaError(hipMalloc(&d_out, out_size));

    checkCudaError(hipMemcpy(d_lhs, h_lhs, lhs_size, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_rhs, h_rhs, rhs_size, hipMemcpyHostToDevice));

    /* We assume block dim to be 32 */
    const u32 num_threads = std::min(dim, 32u);
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks = (dim + num_threads - 1) / num_threads;
    const dim3 grid_dims(num_blocks, num_blocks);

    kernel_matmul_tiled_cu<<<grid_dims, block_dims>>>(
        d_lhs,
        d_rhs,
        d_out,
        dim,
        lhs_stride,
        rhs_stride,
        out_stride
    );
    hipDeviceSynchronize();

    checkCudaError(hipMemcpy(h_out, d_out, out_size, hipMemcpyDeviceToHost));

    hipFree(d_out);
    hipFree(d_rhs);
    hipFree(d_lhs);

    checkCudaError(hipGetLastError());

    return 0;
}

static int run_kernel_cu_test_(
    const i64 *h_lhs,
    const i64 *h_rhs,
          i64 *h_out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
          i64 *u_lhs;
    const u32  lhs_size = dim * lhs_stride * sizeof(*u_lhs);

          i64 *u_rhs;
    const u32  rhs_size = dim * rhs_stride * sizeof(*u_rhs);

          i64 *u_out;
    const u32  out_size = dim * out_stride * sizeof(*u_out);

    checkCudaError(hipMallocManaged(&u_lhs, lhs_size));
    checkCudaError(hipMallocManaged(&u_rhs, rhs_size));
    checkCudaError(hipMallocManaged(&u_out, out_size));

    memcpy(u_lhs, h_lhs, lhs_size);
    memcpy(u_rhs, h_rhs, rhs_size);

    const u32 num_threads = 32;
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks = (dim + num_threads - 1) / num_threads;
    const dim3 grid_dims(num_blocks, num_blocks);

    kernel_matmul_cu<<<grid_dims, block_dims>>>(
        u_lhs,
        u_rhs,
        u_out,
        dim,
        lhs_stride,
        rhs_stride,
        out_stride
    );
    hipDeviceSynchronize();

    memcpy(h_out, u_out, out_size);

    hipFree(u_out);
    hipFree(u_rhs);
    hipFree(u_lhs);

    checkCudaError(hipGetLastError());

    return 0;
}

EXTERN_C int run_kernel_cu(
          i64 *h_lhs,
    const u32  lhs_cols,
    const u32  lhs_rows,
    const u32  lhs_stride,

          i64 *h_rhs,
    const u32  rhs_cols,
    const u32  rhs_rows,
    const u32  rhs_stride,

          i64 *h_out,
    const u32  out_cols,
    const u32  out_rows,
    const u32  out_stride,

    cuda_kernel_variant variant
) {
    static std::atomic<u32> printed(0);

    assert(lhs_cols == lhs_rows);
    assert(rhs_cols == rhs_rows);
    assert(out_cols == out_rows);
    assert(lhs_cols == rhs_cols);
    assert(lhs_cols == out_cols);

    if (printed.fetch_or(1, std::memory_order_relaxed) == 0 &&
        strcmp(current_dev().name, "NVIDIA GeForce GTX 970")) {
        fprintf(stderr, CLR_YELLOW "WARN: %s: kernel written for %s, but current device is %s.\n" CLR_RESET,
                __func__, "NVIDIA GeForce GTX 970", current_dev().name);
    }

    std::unique_lock lck(kernel_exec_mtx);

    switch (variant) {
    case cuda_kernel_variant::UMEM:
        return run_kernel_cu_umem_(
            h_lhs,
            h_rhs,
            h_out,
            lhs_cols,
            lhs_stride,
            rhs_stride,
            out_stride
        );

    case cuda_kernel_variant::UMEM_TILED:
        return run_kernel_cu_umem_tiled_(
            h_lhs,
            h_rhs,
            h_out,
            lhs_cols,
            lhs_stride,
            rhs_stride,
            out_stride
        );

    case cuda_kernel_variant::TILED:
        return run_kernel_cu_tiled_(
            h_lhs,
            h_rhs,
            h_out,
            lhs_cols,
            lhs_stride,
            rhs_stride,
            out_stride
        );

    case cuda_kernel_variant::TEST:
        return run_kernel_cu_test_(
            h_lhs,
            h_rhs,
            h_out,
            lhs_cols,
            lhs_stride,
            rhs_stride,
            out_stride
        );
    }

    __builtin_unreachable();
}


/****************************************************************************
 * F32 Kernels
 ****************************************************************************/

/*
 * We assume all matrcies are square and have the same dimensions.
 */
__global__ void kernel_matmul_cu_f32(
    const f32 *lhs,
    const f32 *rhs,
          f32 *out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
        const u32 x = threadIdx.x + blockDim.x * blockIdx.x;
        const u32 y = threadIdx.y + blockDim.y * blockIdx.y;

        const bool out_of_bounds = x >= dim | y >= dim;
        if (out_of_bounds)
            return;

        out[x + y*out_stride] = 0;

        for (u32 i = 0; i < dim; ++i)
            out[x + y*out_stride] += lhs[i + y*lhs_stride] * rhs[x + i*rhs_stride];
}

/*
 * We assume all matrcies are square and have the same dimensions.
 */
__global__ void kernel_matmul_tiled_cu_f32(
    const f32 *lhs,
    const f32 *rhs,
          f32 *out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
    constexpr u32 TILE_SIZE = 32 * 32;

    __shared__ f32 tilea[TILE_SIZE];
    __shared__ f32 tileb[TILE_SIZE];
    __shared__ f32 tilec[TILE_SIZE];

    /* Global coordinates for this thread. */
    const u32 gx = threadIdx.x + blockIdx.x * blockDim.x;
    const u32 gy = threadIdx.y + blockIdx.y * blockDim.y;

    const bool out_of_bounds = gx >= dim | gy >= dim;
    if (out_of_bounds)
        return;

    /* Local tile coordinates for this thread. */
    const u32 tx      = threadIdx.x;
    const u32 ty      = threadIdx.y;
    const u32 tstride = blockDim.x;

    tilec[tx + ty * tstride] = 0;

    /*
     * Row is a loop invariant for tilea.
     * Column is a loop invariant for tileb.
     */
    for (int j = 0; j < dim; j += blockDim.x) {

        /*
         * Load lhs and rhs tiles into shared memory.
         * Each thread loads one element.
         */
        tilea[tx + ty * tstride] = lhs[(tx+j) + gy * lhs_stride];
        tileb[tx + ty * tstride] = rhs[gx + (ty+j) * rhs_stride];

        /* Make sure tilea and tileb are populated. */
        __syncthreads();

        /* Accumulate results for current tile. */
        for (u32 i = 0; i < blockDim.x; ++i)
            tilec[tx + ty * tstride] += tilea[i + ty * tstride] * tileb[tx + i * tstride];

        /* Make sure we don't modify tilea and tileb before all threads are finihsed with per tile computation */
        __syncthreads();
    }

    out[gx + gy * out_stride] = tilec[tx + ty * tstride];
}

/*
 * We assume all matrcies are square and have the same dimensions.
 */
__global__ void kernel_matmul_test_cu_f32(
    const f32 *lhs,
    const f32 *rhs,
          f32 *out_,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
    /* Global coordinates for this thread. */
    const u32 gx = blockIdx.z + blockIdx.x * blockDim.x;
    const u32 gy = blockIdx.z + blockIdx.y * blockDim.y;

    const bool out_of_bounds = gx >= dim | gy >= dim;
    if (out_of_bounds)
        return;

    u32 out = 0;

    for (u32 i = 0; i < dim; ++i)
        out += lhs[i + gy * lhs_stride] * rhs[gx + i * rhs_stride];

    out_[gx + gy * out_stride] = out;
}

static int run_kernel_cu_umem_f32_(
    const f32 *h_lhs,
    const f32 *h_rhs,
          f32 *h_out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
          f32 *u_lhs;
    const u32  lhs_size = dim * lhs_stride * sizeof(*u_lhs);

          f32 *u_rhs;
    const u32  rhs_size = dim * rhs_stride * sizeof(*u_rhs);

          f32 *u_out;
    const u32  out_size = dim * out_stride * sizeof(*u_out);

    checkCudaError(hipMallocManaged(&u_lhs, lhs_size));
    checkCudaError(hipMallocManaged(&u_rhs, rhs_size));
    checkCudaError(hipMallocManaged(&u_out, out_size));

    memcpy(u_lhs, h_lhs, lhs_size);
    memcpy(u_rhs, h_rhs, rhs_size);

    /* We assume block dim to be 32 */
    const u32 num_threads = std::min(dim, 32u);
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks = (dim + 31u) / 32u;
    const dim3 grid_dims(num_blocks, num_blocks);

    kernel_matmul_cu_f32<<<grid_dims, block_dims>>>(
        u_lhs,
        u_rhs,
        u_out,
        dim,
        lhs_stride,
        rhs_stride,
        out_stride
    );
    hipDeviceSynchronize();

    memcpy(h_out, u_out, out_size);

    hipFree(u_out);
    hipFree(u_rhs);
    hipFree(u_lhs);

    checkCudaError(hipGetLastError());

    return 0;
}

static int run_kernel_cu_umem_tiled_f32_(
    const f32 *h_lhs,
    const f32 *h_rhs,
          f32 *h_out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
          f32 *u_lhs;
    const u32  lhs_size = dim * lhs_stride * sizeof(*u_lhs);

          f32 *u_rhs;
    const u32  rhs_size = dim * rhs_stride * sizeof(*u_rhs);

          f32 *u_out;
    const u32  out_size = dim * out_stride * sizeof(*u_out);

    checkCudaError(hipMallocManaged(&u_lhs, lhs_size));
    checkCudaError(hipMallocManaged(&u_rhs, rhs_size));
    checkCudaError(hipMallocManaged(&u_out, out_size));

    memcpy(u_lhs, h_lhs, lhs_size);
    memcpy(u_rhs, h_rhs, rhs_size);

    /* We assume block dim to be 32 */
    const u32 num_threads = std::min(dim, 32u);
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks = (dim + num_threads - 1) / num_threads;
    const dim3 grid_dims(num_blocks, num_blocks);

    kernel_matmul_tiled_cu_f32<<<grid_dims, block_dims>>>(
        u_lhs,
        u_rhs,
        u_out,
        dim,
        lhs_stride,
        rhs_stride,
        out_stride
    );
    hipDeviceSynchronize();

    memcpy(h_out, u_out, out_size);

    hipFree(u_out);
    hipFree(u_rhs);
    hipFree(u_lhs);

    checkCudaError(hipGetLastError());

    return 0;
}

static int run_kernel_cu_tiled_f32_(
    const f32 *h_lhs,
    const f32 *h_rhs,
          f32 *h_out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
          f32 *d_lhs;
    const u32  lhs_size = dim * lhs_stride * sizeof(*d_lhs);

          f32 *d_rhs;
    const u32  rhs_size = dim * rhs_stride * sizeof(*d_rhs);

          f32 *d_out;
    const u32  out_size = dim * out_stride * sizeof(*d_out);

    checkCudaError(hipMalloc(&d_lhs, lhs_size));
    checkCudaError(hipMalloc(&d_rhs, rhs_size));
    checkCudaError(hipMalloc(&d_out, out_size));

    checkCudaError(hipMemcpy(d_lhs, h_lhs, lhs_size, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_rhs, h_rhs, rhs_size, hipMemcpyHostToDevice));

    /* We assume block dim to be 32 */
    const u32 num_threads = std::min(dim, 32u);
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks = (dim + num_threads - 1) / num_threads;
    const dim3 grid_dims(num_blocks, num_blocks);

    kernel_matmul_tiled_cu_f32<<<grid_dims, block_dims>>>(
        d_lhs,
        d_rhs,
        d_out,
        dim,
        lhs_stride,
        rhs_stride,
        out_stride
    );
    hipDeviceSynchronize();

    checkCudaError(hipMemcpy(h_out, d_out, out_size, hipMemcpyDeviceToHost));

    hipFree(d_out);
    hipFree(d_rhs);
    hipFree(d_lhs);

    checkCudaError(hipGetLastError());

    return 0;
}

static int run_kernel_cu_test_f32_(
    const f32 *h_lhs,
    const f32 *h_rhs,
          f32 *h_out,
    const u32  dim,
    const u32  lhs_stride,
    const u32  rhs_stride,
    const u32  out_stride
) {
          f32 *u_lhs;
    const u32  lhs_size = dim * lhs_stride * sizeof(*u_lhs);

          f32 *u_rhs;
    const u32  rhs_size = dim * rhs_stride * sizeof(*u_rhs);

          f32 *u_out;
    const u32  out_size = dim * out_stride * sizeof(*u_out);

    checkCudaError(hipMallocManaged(&u_lhs, lhs_size));
    checkCudaError(hipMallocManaged(&u_rhs, rhs_size));
    checkCudaError(hipMallocManaged(&u_out, out_size));

    memcpy(u_lhs, h_lhs, lhs_size);
    memcpy(u_rhs, h_rhs, rhs_size);

    const u32 num_threads = 32;
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks = (dim + num_threads - 1) / num_threads;
    const dim3 grid_dims(num_blocks, num_blocks);

    kernel_matmul_cu_f32<<<grid_dims, block_dims>>>(
        u_lhs,
        u_rhs,
        u_out,
        dim,
        lhs_stride,
        rhs_stride,
        out_stride
    );
    hipDeviceSynchronize();

    memcpy(h_out, u_out, out_size);

    hipFree(u_out);
    hipFree(u_rhs);
    hipFree(u_lhs);

    checkCudaError(hipGetLastError());

    return 0;
}

EXTERN_C int run_kernel_cu_f32(
          f32 *h_lhs,
    const u32  lhs_cols,
    const u32  lhs_rows,
    const u32  lhs_stride,

          f32 *h_rhs,
    const u32  rhs_cols,
    const u32  rhs_rows,
    const u32  rhs_stride,

          f32 *h_out,
    const u32  out_cols,
    const u32  out_rows,
    const u32  out_stride,

    cuda_kernel_variant variant
) {
    static std::atomic<u32> printed(0);

    assert(lhs_cols == lhs_rows);
    assert(rhs_cols == rhs_rows);
    assert(out_cols == out_rows);
    assert(lhs_cols == rhs_cols);
    assert(lhs_cols == out_cols);

    if (printed.fetch_or(1, std::memory_order_relaxed) == 0 &&
        strcmp(current_dev().name, "NVIDIA GeForce GTX 970")) {
        fprintf(stderr, CLR_YELLOW "WARN: %s: kernel written for %s, but current device is %s.\n" CLR_RESET,
                __func__, "NVIDIA GeForce GTX 970", current_dev().name);
    }

    std::unique_lock lck(kernel_exec_mtx);

    switch (variant) {
    case cuda_kernel_variant::UMEM:
        return run_kernel_cu_umem_f32_(
            h_lhs,
            h_rhs,
            h_out,
            lhs_cols,
            lhs_stride,
            rhs_stride,
            out_stride
        );

    case cuda_kernel_variant::UMEM_TILED:
        return run_kernel_cu_umem_tiled_f32_(
            h_lhs,
            h_rhs,
            h_out,
            lhs_cols,
            lhs_stride,
            rhs_stride,
            out_stride
        );

    case cuda_kernel_variant::TILED:
        return run_kernel_cu_tiled_f32_(
            h_lhs,
            h_rhs,
            h_out,
            lhs_cols,
            lhs_stride,
            rhs_stride,
            out_stride
        );

    case cuda_kernel_variant::TEST:
        return run_kernel_cu_test_f32_(
            h_lhs,
            h_rhs,
            h_out,
            lhs_cols,
            lhs_stride,
            rhs_stride,
            out_stride
        );
    }

    __builtin_unreachable();
}

struct matrix {
    constexpr static u32 STRIDE_AUTO = 0;

    enum class grad_t {
        no_grad,
        with_grad,
    };

    void free()
    {
        if (this->data)
            hipFree(this->data);

        if (this->grad)
            hipFree(this->grad);

        this->data = nullptr;
        this->grad = nullptr;
        this->width = 0;
        this->height = 0;
        this->stride = 0;
    }

    hipError_t alloc_on_device(const matview_f32_t& mat, grad_t grad_opt)
    {
        return this->alloc_on_device(mat.width, mat.height, mat.stride, grad_opt);
    }

    hipError_t alloc_on_device(u32 width, u32 height, u32 stride, grad_t grad_opt)
    {
        this->free();

        if (stride == STRIDE_AUTO)
            stride = width;

        assert(width <= stride);

        auto err = hipMalloc(&this->data, sizeof(*this->data) * height * stride);
        if (err != hipSuccess)
            return err;

        if (grad_opt == grad_t::with_grad) {
            err = hipMalloc(&this->grad, sizeof(*this->grad) * height * stride);
            if (err != hipSuccess) {
                hipFree(this->data);
                this->data = nullptr;
                return err;
            }
        }

        this->width = width;
        this->height = height;
        this->stride = stride;

        return hipSuccess;
    }

    u64 size_bytes() const
    {
        return this->height * this->stride * sizeof(*this->data);
    }

    bool has_grad() const
    {
        return this->grad != nullptr;
    }

    f32 *data = nullptr;
    f32 *grad = nullptr;
    u32  width = 0;
    u32  height = 0;
    u32  stride = 0;
};

constexpr static auto STRIDE_AUTO = matrix::STRIDE_AUTO;

constexpr static auto no_grad   = matrix::grad_t::no_grad;
constexpr static auto with_grad = matrix::grad_t::with_grad;

/*
 * Computes:
 *     out = lhs @ rhs
 *
 *  Assumes 'lhs', 'rhs' and 'out' are correctly sized.
 */
__global__ void kernel_matmul(
    struct matrix lhs,
    struct matrix rhs,
    struct matrix out
) {
    const u32 gx = threadIdx.x + blockIdx.x * blockDim.x;
    const u32 gy = threadIdx.y + blockIdx.y * blockDim.y;

    const bool out_of_bounds = (gx >= out.width) | (gy >= out.height);
    if (out_of_bounds)
        return;

    f32 v = 0;
    for (u32 i = 0; i < lhs.width; ++i)
        v += lhs.data[i + gy * lhs.stride] * rhs.data[gx + i * rhs.stride];
    out.data[gx + gy * out.stride] = v;
}

__global__ void kernel_mse(
    struct matrix lhs,
    struct matrix rhs,
    struct matrix out_mse,
    f32 *out_mse_v
){
    const u32 gx = threadIdx.x + blockIdx.x * blockDim.x;
    const u32 gy = threadIdx.y + blockIdx.y * blockDim.y;

    const bool out_of_bounds = (gx >= out_mse.width) | (gy >= out_mse.height);
    if (out_of_bounds)
        return;

    /*
     * Compute:
     *     data = (lhs - rhs) ** 2
     *     grad = 2 * (lhs - rhs)
     */
    f32 mse = lhs.data[gx + gy * lhs.stride] - rhs.data[gx + gy * rhs.stride];
    out_mse.grad[gx + gy * out_mse.stride] = 2*mse;
    out_mse.data[gx + gy * out_mse.stride] = mse * mse;

    __syncthreads();

    /*
     * From now on we execute:
     *     num_vectors = lhs.width = rhs.width
     *     x: [0; num_vectors]
     *     y: 0
     *
     *  IOW reduce across columns.
     */
    if (gy != 0)
        return;

    f32 mse_v = 0;
    for (u32 y = 0; y < out_mse.height; ++y)
        mse_v += out_mse.data[gx + y * out_mse.stride];

    *out_mse_v = mse_v;
}

__global__ void kernel_grad_cu_forward(
    struct matrix xs,
    struct matrix ys,
    struct matrix ypred,
    struct matrix w,
    struct matrix loss0,
    struct matrix loss1,
    f32 *out_loss_v
){
    const u32 gx = threadIdx.x;
    const u32 gy = threadIdx.y;

    const bool out_of_bounds = (gx >= ypred.width) | (gy >= ypred.height);

    if (out_of_bounds)
        return;

    __syncthreads();

    if (gx != 0 | gy != 0)
        return;

    f32 loss_v = 0;
    for (u32 i = 0; i < loss1.height; ++i)
        loss_v += loss1.data[gx + i * loss1.stride];

    *out_loss_v = loss_v;
}

__global__ void kernel_grad_reset(struct matrix m)
{
    const u32 gx = threadIdx.x;
    const u32 gy = threadIdx.y;

    const bool out_of_bounds = (gx >= m.width) | (gy >= m.height);
    if (out_of_bounds)
        return;

    m.grad[gx + gy * m.stride] = 1.0f;
}

__global__ void kernel_backward_x_squared(
    struct matrix cur,
    struct matrix prev
) {
    const u32 gx = threadIdx.x;
    const u32 gy = threadIdx.y;

    const bool out_of_bounds = (gx >= cur.width) | (gy >= cur.height);
    if (out_of_bounds)
        return;

    prev.grad[gx + gy * prev.stride] *= 2*cur.grad[gx + gy * cur.stride];
}

__global__ void kernel_grad_cu_backward(
    struct matrix xs,
    struct matrix w,
    struct matrix loss
){
    const f32 lr = 1e-6f;

    const u32 gx = threadIdx.x;
    const u32 gy = threadIdx.y;

    const bool out_of_bounds = (gx >= w.width) | (gy >= w.height);
    if (out_of_bounds)
        return;

    w.grad[gx + gy * w.stride] = loss.grad[0 + gy * loss.stride] * xs.data[0 + gx * xs.stride];
    w.data[gx + gy * w.stride] -= w.grad[gx + gy * w.stride] * lr;
}

template<typename FloatType>
FloatType infinity_of(FloatType x)
{
    (void) x;
    return std::numeric_limits<std::remove_reference_t<FloatType>>::infinity();
}

EXTERN_C void run_kernel_cu_grad_f32(
    matview_f32_t h_xs,
    matview_f32_t h_ygt,
    mat_f32_t &h_weights,
    f32 &out_loss
) {
    assert(h_xs.width == h_ygt.width);
    assert(h_xs.height == h_ygt.height);
    assert(h_xs.width == 1);
    assert(h_ygt.width == 1);

    struct matrix d_xs, d_ygt, d_ypred, d_weights, d_loss;
    f32 *d_lossv;
    const u32 num_neurons = h_xs.height;

    checkCudaError(d_xs.alloc_on_device(h_xs.width, h_xs.height, h_xs.stride, no_grad));
    checkCudaError(d_ygt.alloc_on_device(d_ygt.width, h_ygt.height, h_ygt.stride, no_grad));
    checkCudaError(d_ypred.alloc_on_device(h_ygt.width, h_ygt.height, h_ygt.stride, no_grad));
    checkCudaError(d_weights.alloc_on_device(h_xs.height, num_neurons, STRIDE_AUTO, with_grad));
    checkCudaError(d_loss.alloc_on_device(d_ypred.width, d_ypred.height, d_ypred.stride, with_grad));
    checkCudaError(hipMalloc(&d_lossv, sizeof(*d_lossv)));

    mat_f32_t h_grad = mat_f32_t::make_matrix_zero(d_weights.width, d_weights.height, d_weights.stride);
    h_weights = mat_f32_t::make_matrix_zero(d_weights.width, d_weights.height, d_weights.stride);

    assert(d_xs.stride == h_xs.stride);
    assert(d_xs.size_bytes() == h_xs.size_bytes());
    checkCudaError(hipMemcpy(d_xs.data, h_xs.data, d_xs.size_bytes(), hipMemcpyHostToDevice));

    assert(d_ygt.stride == h_ygt.stride);
    assert(d_ygt.size_bytes() == h_ygt.size_bytes());
    checkCudaError(hipMemcpy(d_ygt.data, h_ygt.data, d_ygt.size_bytes(), hipMemcpyHostToDevice));

    assert(d_weights.size_bytes() == h_weights.size_bytes());
    checkCudaError(hipMemcpy(d_weights.data, h_weights.data.get(), d_weights.size_bytes(), hipMemcpyHostToDevice));


    const u32 num_threads = 32;
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks = 1;
    const dim3 grid_dims(num_blocks, num_blocks);

    constexpr f32 LOSS_TARGET = 1e-5f;
    out_loss = infinity_of(out_loss);
    u32 epoch = 0;
    while(!should_exit() && out_loss > LOSS_TARGET) {

        /*
         * Forward pass
         */
        kernel_matmul<<<grid_dims, block_dims>>>(
            d_weights,
            d_xs,
            d_ypred
        );

        kernel_mse<<<grid_dims, block_dims>>>(
            d_ypred,
            d_ygt,
            d_loss,
            d_lossv
        );

        checkCudaError(hipMemcpy(&out_loss, d_lossv, sizeof(f32), hipMemcpyDeviceToHost));

        kernel_grad_cu_backward<<<grid_dims, block_dims>>>(
            d_xs,
            d_weights,
            d_loss
        );

        checkCudaError(hipMemcpy(h_grad.data.get(), d_weights.grad, d_weights.size_bytes(),
                                  hipMemcpyDeviceToHost));

        /* Copy trained weights back to the caller. */
        checkCudaError(hipMemcpy(h_weights.data.get(), d_weights.data, d_weights.size_bytes(),
                                  hipMemcpyDeviceToHost));

        /* Copy loss back to the caller. */
        checkCudaError(hipMemcpy(&out_loss, d_lossv, sizeof(f32),
                                  hipMemcpyDeviceToHost));

        if (++epoch % 1024 == 0 || out_loss <= LOSS_TARGET) {
            puts("grad");
            print_mat(h_grad);

            puts("weights");
            print_mat(h_weights);

            auto ypred = mat_mul_cpu(h_weights, h_xs);
            puts("ypred");
            print_mat(ypred);

            printf("loss: %f\n", out_loss);
        }
    }

    hipFree(d_lossv);
    d_loss.free();
    d_weights.free();
    d_ypred.free();
    d_ygt.free();
    d_xs.free();
}

/* Classify learning rate. */
static f32 LEARNING_RATE = 1e-10;

__global__ void kernel_hidden_forward(
    struct matrix lhs,
    struct matrix rhs,
    struct matrix out
) {
    const u32 gx = threadIdx.x + blockIdx.x * blockDim.x;
    const u32 gy = threadIdx.y + blockIdx.y * blockDim.y;

    const bool out_of_bounds = (gx >= out.width) | (gy >= out.height);
    if (out_of_bounds)
        return;

    /* Compute forward pass. */
    f32 v = 0;
    for (u32 i = 0; i < lhs.height; ++i)
        v += lhs.data[gx + i * lhs.stride] * rhs.data[i + gy * rhs.stride];
    out.data[gx + gy * out.stride] = v;
}

static void hidden_forward(
    struct matrix lhs,
    struct matrix rhs,
    struct matrix out
) {
    const u32 num_threads = 32;
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks_x = (out.width  + num_threads-1u) / num_threads;
    const u32 num_blocks_y = (out.height + num_threads-1u) / num_threads;
    const dim3 grid_dims(num_blocks_x, num_blocks_y);

    kernel_hidden_forward<<<grid_dims, block_dims>>>(lhs, rhs, out);
}

__global__ void kernel_mse_reduce(
    struct matrix ypred,
    struct matrix ygt,
    f32 *out_loss_v
) {
    const u32 nblock  = blockIdx.y;
    const u32 blocksz = blockDim.x * blockDim.y;
    const u32 gid     = (nblock * blocksz) + (threadIdx.x + blockDim.x * threadIdx.y);

    const bool out_of_bounds = gid >= ypred.height;
    if (out_of_bounds)
        return;

    const f32 v    = ypred.data[0 + gid * ypred.stride];
    const f32 diff = v - ygt.data[0 + gid * ygt.stride];
    const f32 mse  = diff * diff;

    ypred.data[0 + gid * ypred.stride] = mse;

    /* Compute gradient right away - it's the last step. */
    ypred.grad[0 + gid * ypred.stride] = 2*v;

    __syncthreads();

    if (gid != 0)
        return;

    f32 sum = 0.0f;
    for(u32 i = 0; i < ypred.height; ++i)
        sum += ypred.data[0 + i * ypred.stride];

    *out_loss_v = sum;
}

static void mse_reduce(
    struct matrix ypred,
    struct matrix ygt,
    f32 *out_loss_v
) {
    assert(ypred.width == 1);

    const u32 num_threads = 32;
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks_y = (ypred.height+1023u) / 1024u;
    const dim3 grid_dims(1, num_blocks_y);

    kernel_mse_reduce<<<grid_dims, block_dims>>>(ypred, ygt, out_loss_v);
}

__global__ void kernel_hidden_backward_hidden(
    struct matrix lhs,
    struct matrix rhs,
    struct matrix parent
) {
    const u32 gx = threadIdx.x + blockIdx.x * blockDim.x;
    const u32 gy = threadIdx.y + blockIdx.y * blockDim.y;

    const bool out_of_bounds = (gx >= lhs.width) | (gy >= lhs.height);
    if (out_of_bounds)
        return;

    f32 v = 0.0;
    for (u32 i = 0; i < rhs.height; ++i)
        v += parent.grad[gx + i * parent.stride] * rhs.data[gy + i * rhs.stride];

    lhs.grad[gx + gy * lhs.stride] = v;
}

__global__ void kernel_hidden_backward_data(
    struct matrix lhs,
    struct matrix rhs,
    struct matrix parent
) {
    const u32 gx = threadIdx.x + blockIdx.x * blockDim.x;
    const u32 gy = threadIdx.y + blockIdx.y * blockDim.y;

    const bool out_of_bounds = (gx >= rhs.width) | (gy >= rhs.height);
    if (out_of_bounds)
        return;

    f32 v = 0.0;
    for (u32 i = 0; i < lhs.width; ++i)
        v += parent.grad[i + gy * parent.stride] * lhs.data[i + gx * lhs.stride];

    rhs.grad[gx + gy * rhs.stride] = v;
}

static void hidden_backward(
    struct matrix lhs,
    struct matrix rhs,
    struct matrix parent
) {
    assert(lhs.has_grad());
    assert(parent.has_grad());

    const u32 num_threads = 32;
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks_hidden_x = (lhs.width  + num_threads-1u) / num_threads;
    const u32 num_blocks_hidden_y = (lhs.height + num_threads-1u) / num_threads;
    const dim3 grid_dims_w(num_blocks_hidden_x, num_blocks_hidden_y);

    assert(rhs.height == parent.height);
    assert(lhs.width  == parent.width);
    assert(lhs.height == rhs.width);
    kernel_hidden_backward_hidden<<<grid_dims_w, block_dims>>>(lhs, rhs, parent);

    if (!rhs.has_grad())
        return;

    const u32 num_blocks_data_x = (rhs.width  + num_threads-1u) / num_threads;
    const u32 num_blocks_data_y = (rhs.height + num_threads-1u) / num_threads;
    const dim3 grid_dims_d(num_blocks_data_x, num_blocks_data_y);

    assert(lhs.width  == parent.width);
    assert(rhs.width  == lhs.height);
    assert(rhs.height == parent.height);
    kernel_hidden_backward_data<<<grid_dims_d, block_dims>>>(lhs, rhs, parent);
}

__global__ void kernel_apply_gradient(struct matrix m, f32 lr)
{
    const u32 gx = threadIdx.x + blockIdx.x * blockDim.x;
    const u32 gy = threadIdx.y + blockIdx.y * blockDim.y;

    const bool out_of_bounds = (gx >= m.width) | (gy >= m.height);
    if (out_of_bounds)
        return;

    m.data[gx + gy * m.stride] -= m.grad[gx + gy * m.stride] * lr;
}

static void apply_gradient(struct matrix m)
{
    assert(m.has_grad());

    const u32 num_threads = 32;
    const dim3 block_dims(num_threads, num_threads);

    const u32 num_blocks_x = (m.width  + num_threads-1u) / num_threads;
    const u32 num_blocks_y = (m.height + num_threads-1u) / num_threads;
    const dim3 grid_dims(num_blocks_x, num_blocks_y);

    kernel_apply_gradient<<<grid_dims, block_dims>>>(m, LEARNING_RATE);
}

/* Print human-readable duration. */
static void print_human_duration(std::chrono::seconds duration)
{
    using namespace std::chrono;

    int total_seconds = duration.count();
    int hours = total_seconds / 3600;
    int minutes = (total_seconds % 3600) / 60;
    int seconds = total_seconds % 60;

    if (hours > 0) {
        printf("%dh %dm %ds", hours, minutes, seconds);
    } else if (minutes > 0) {
        printf("%dm %ds", minutes, seconds);
    } else {
        printf("%ds", seconds);
    }
}

EXTERN_C void train_cu_classify(
    matview_f32_t h_xs,
    matview_f32_t h_ygt,
    std::array<mat_f32_t, 3> &h_weights,
    f32 &out_loss
) {
    constexpr u32 num_epochs = 1024u * 128u;
    constexpr u32 num_layers = 3u;
    const u32 params[] = {h_xs.width, 10u, 8u, 1u}; /* Number of parameters for each layer. */

    struct matrix d_xs, d_ygt, d_hidden[num_layers], d_ypred[num_layers];
    mat_f32_t h_ypred[num_layers];
    f32 *d_lossv;

    /* For debug. */
    constexpr bool debug = false;
    mat_f32_t h_hidden_grads[num_layers];
    mat_f32_t h_ypred_grads[num_layers];

    /* Allocate buffers for host inputs. */
    checkCudaError(d_xs.alloc_on_device(h_xs, no_grad));
    checkCudaError(d_ygt.alloc_on_device(h_ygt, no_grad));

    /*
     * Hidden layers. Last one reduces the output to a single scalar for classification.
     *
     * [10;  2] x [ 2; 4096] => [10; 4096]
     * [ 8; 10] x [10; 4096] => [ 8; 4096]
     * [ 1;  8] x [ 8; 4096] => [ 1; 4096]
     *
     */
    for (u32 i = 0; i < num_layers; ++i)
        checkCudaError(d_hidden[i].alloc_on_device(params[i+1], params[i], STRIDE_AUTO, with_grad));

    /* Buffers for holding results after forward pass through hidden layers. */
    for (u32 i = 0; i < num_layers; ++i)
        checkCudaError(d_ypred[i].alloc_on_device(params[i+1], d_xs.height, STRIDE_AUTO, with_grad));

    for (u32 i = 0; i < std::size(d_ypred); ++i) {
        h_ypred[i]       = mat_f32_t::make_matrix(d_ypred[i].width, d_ypred[i].height, d_ypred[i].stride);
        h_ypred_grads[i] = mat_f32_t::make_matrix(d_ypred[i].width, d_ypred[i].height, d_ypred[i].stride);
    }

    /* Scalar describing a loss that we copy back to the caller. */
    checkCudaError(hipMalloc(&d_lossv, sizeof(*d_lossv)));

    checkCudaError(hipMemcpy( d_xs.data,  h_xs.data,  d_xs.size_bytes(), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_ygt.data, h_ygt.data, d_ygt.size_bytes(), hipMemcpyDeviceToHost));

    for (u32 i = 0; i < std::size(d_hidden); ++i) {
        h_weights[i] = mat_f32_t::make_matrix_in_range(
            d_hidden[i].width,
            d_hidden[i].height,
            d_hidden[i].stride,
            -1.0f, 1.0f
        );
        h_hidden_grads[i] = mat_f32_t::make_matrix(
            d_hidden[i].width,
            d_hidden[i].height,
            d_hidden[i].stride
        );
    }

    for (u32 i = 0; i < std::size(d_hidden); ++i) {
        assert(d_hidden[i].size_bytes() == h_weights[i].size_bytes());
        checkCudaError(hipMemcpy(d_hidden[i].data, h_weights[i].data.get(), d_hidden[i].size_bytes(),
                                  hipMemcpyHostToDevice));
    }

    /* This time measuring perhaps has some problems with TSC wrap-around. IDC ;] */
    using ticker_t  = std::chrono::high_resolution_clock;
    using std::chrono::seconds;
    using std::chrono::milliseconds;
    using std::chrono::duration_cast;
    auto status_start       = ticker_t::now();
    auto status_last_update = status_start;
    auto loss_prev = std::numeric_limits<f32>::infinity();

    auto maybe_print_status_line = [&] (const auto epoch, const auto max_epochs, bool force = false) {
        f32 loss;

        const auto now = ticker_t::now();
        if (!force && (now - status_last_update < milliseconds(250)))
            return;

        status_last_update = now;

        checkCudaError(hipMemcpy(&loss, d_lossv, sizeof(f32),
                       hipMemcpyDeviceToHost));

        printf(
            "\033[2K\r" /* Clear line, carriage return. */
            "epochs: %u/%u, loss: %f, lr: %.01e, duration: ",
            epoch, max_epochs, loss, LEARNING_RATE
        );

        if (loss > loss_prev)
            LEARNING_RATE /= 10.0f;

        loss_prev = loss;

        print_human_duration(duration_cast<seconds>(now - status_start));
        fflush(stdout);
    };

    /* Train. */
    u32 epoch;
    for (epoch = 0; epoch < num_epochs && (!should_exit()); ++epoch) {
        hidden_forward(d_hidden[0],       d_xs, d_ypred[0]);
        hidden_forward(d_hidden[1], d_ypred[0], d_ypred[1]);
        hidden_forward(d_hidden[2], d_ypred[1], d_ypred[2]);
        mse_reduce(d_ypred[2], d_ygt, d_lossv);
        hidden_backward(d_hidden[2], d_ypred[1], d_ypred[2]);
        hidden_backward(d_hidden[1], d_ypred[0], d_ypred[1]);
        hidden_backward(d_hidden[0],       d_xs, d_ypred[0]);
        apply_gradient(d_hidden[2]);
        apply_gradient(d_hidden[1]);
        apply_gradient(d_hidden[0]);

        maybe_print_status_line(epoch, num_epochs);
    }
    maybe_print_status_line(epoch, num_epochs, true);
    putchar('\n');

    /* Copy hidden layers to host memory. */
    for (u32 i = 0; i < std::size(d_hidden); ++i) {
        assert(      d_hidden[i].size_bytes() == h_weights[i].size_bytes());
        assert(h_hidden_grads[i].size_bytes() == h_weights[i].size_bytes());
        checkCudaError(hipMemcpy(h_weights[i].data.get(), d_hidden[i].data, d_hidden[i].size_bytes(),
                                  hipMemcpyDeviceToHost));
        checkCudaError(hipMemcpy(h_hidden_grads[i].data.get(), d_hidden[i].grad, d_hidden[i].size_bytes(),
                                  hipMemcpyDeviceToHost));
    }

    /* Copy ypred partial results to host memory and caller. For debug. */
    if (debug) {
        for (u32 i = 0; i < std::size(d_ypred); ++i) {
            assert(      d_ypred[i].size_bytes() == h_ypred[i].size_bytes());
            assert(h_ypred_grads[i].size_bytes() == h_ypred[i].size_bytes());
            checkCudaError(hipMemcpy(h_ypred[i].data.get(), d_ypred[i].data, d_ypred[i].size_bytes(),
                                      hipMemcpyDeviceToHost));
            checkCudaError(hipMemcpy(h_ypred_grads[i].data.get(), d_ypred[i].grad, d_ypred[i].size_bytes(),
                                      hipMemcpyDeviceToHost));
        }
    }

    if (debug) {
        print_mat(    "hidden0", h_weights[0]);
        print_mat(      "grad0", h_hidden_grads[0]);
        print_mat(    "hidden1", h_weights[1]);
        print_mat(      "grad1", h_hidden_grads[1]);
        print_mat(    "hidden2", h_weights[2]);
        print_mat(      "grad2", h_hidden_grads[2]);
        print_mat(     "ypred2", h_ypred[2]);
        print_mat("ypred_grad0", h_ypred_grads[0]);
        print_mat("ypred_grad1", h_ypred_grads[1]);
        print_mat("ypred_grad2", h_ypred_grads[2]);
    }

    checkCudaError(hipMemcpy(&out_loss, d_lossv, sizeof(f32),
                   hipMemcpyDeviceToHost));

    hipFree(d_lossv);
    for (auto &m: d_ypred)  m.free();
    for (auto &m: d_hidden) m.free();
    d_ygt.free();
    d_xs.free();
}

